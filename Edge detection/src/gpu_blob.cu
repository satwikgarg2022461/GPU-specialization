#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#include <cstddef>
#include <exception>
#include <iostream>
#include <stdexcept>
#include <string>

#include "gpu_blob.hpp"

GpuBlob::GpuBlob(std::size_t size) : m_size(size), m_data(nullptr) {
    const hipError_t err = hipMalloc(&m_data, size);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate device memory at " + std::string(__FILE__) +
                                 ":" + std::to_string(__LINE__));
    }
}
GpuBlob::~GpuBlob() {
    const hipError_t err = hipFree(m_data);
    if (err != hipSuccess) {
        std::cerr << "Failed to free device memory at " << __FILE__ << ":" << __LINE__ << '\n';
        std::terminate();
    }
}
void GpuBlob::copyFrom(const void* data) {
    const hipError_t err = hipMemcpy(m_data, data, m_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data to device at " + std::string(__FILE__) + ":" +
                                 std::to_string(__LINE__));
    }
}
void GpuBlob::copyTo(void* data) const {
    const hipError_t err = hipMemcpy(data, m_data, m_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data from device at " + std::string(__FILE__) +
                                 ":" + std::to_string(__LINE__));
    }
}
void* GpuBlob::data() { return m_data; }
const void* GpuBlob::data() const { return m_data; }
std::size_t GpuBlob::size() const { return m_size; }
